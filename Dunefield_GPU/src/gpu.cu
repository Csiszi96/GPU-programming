#include "gpu.h"

// template <typename T>
// void gpu_malloc(T src, int size) {
//     hipError_t err = hipSuccess;
//     err = hipMalloc(src , size);
//     if( err != hipSuccess){ 
//         std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; 
//         exit(-1); 
//     }
// }

// template <typename T1, typename T2>
// void gpu_copy(T1 src, T2 data, int size, hipMemcpyKind direction) {
//     hipError_t err = hipSuccess;
// 	err = hipMemcpy(src, data, size, direction );
//     if( err != hipSuccess){
//         if (direction == hipMemcpyHostToDevice) std::cout << "Error copying memory from host to device: ";
//         if (direction == hipMemcpyDeviceToHost) std::cout << "Error copying memory from device to host: ";
//         if (direction == hipMemcpyDeviceToDevice) std::cout << "Error copying memory from device to device: ";
//         if (direction == hipMemcpyHostToHost) std::cout << "Error copying memory from host to host: ";
        
//         std::cout << hipGetErrorString(err) << "\n"; 
//         exit(-1); 
//     }
// }

// template <typename T>
// void gpu_free(T src) {
//     hipError_t err = hipSuccess;
//     err = hipFree(src);
//     if( err != hipSuccess) {
//         std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; 
//         exit(-1); 
//     }
// }

// template <typename T>
// void gpu_free(T src, std::string name) {
//     hipError_t err = hipSuccess;
//     err = hipFree(src);
//     if( err != hipSuccess) {
//         std::cout 
//             << "Error freeing allocation for " << name << ": " 
//             << hipGetErrorString(err) << "\n"; 
//         exit(-1); 
//     }
// }


void gpu_check_err(std::string message) {
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if( err != hipSuccess) {
        std::cout 
            << message << hipGetErrorString(err) << "\n"; 
        exit(-1); 
    }
}

void gpu_stream_create(hipStream_t &stream) {
    hipError_t err = hipSuccess;
    err = hipStreamCreate(&stream);
    if( err != hipSuccess) {
        std::cout 
            << "Error creating CUDA stream: " << hipGetErrorString(err) << "\n"; 
        exit(-1); 
    }
}

void gpu_event_create(hipEvent_t &event) {
    hipError_t err = hipSuccess;
    err = hipEventCreate(&event);
    if( err != hipSuccess) {
        std::cout 
            << "Error creating CUDA event: " << hipGetErrorString(err) << "\n"; 
        exit(-1); 
    }
}

void gpu_event(std::string event, hipEvent_t &evt, hipStream_t &stream) {
    hipError_t err = hipSuccess;
    err = hipEventRecord(evt, stream);
    if( err != hipSuccess) {
        std::cout 
            << "Error recording event" << event << ": " << hipGetErrorString(err) << "\n"; 
        exit(-1); 
    }
}